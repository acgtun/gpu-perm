#include "hip/hip_runtime.h"
#include "alloc_kernel.h"

__global__ void gpu_kernel(const Option * opt, const RefGenome * refGenome,
		const HashTable * hashTable, ReadsMatch * readsMatch,
		ResultMatchedReads * result) {
	SIZE_T i = threadIdx.x + blockDim.x * blockIdx.x;
	printf("i = %d\n", i);
//	printf("readsMatch->nReads = %d\n", readsMatch->nReads);
//	if (i >= readsMatch->nReads)
//		return;

	ResultMatchedReads oneResult;
	MappingOneRead(&opt->matchOpt, refGenome, hashTable,
			readsMatch->readsInBits[i], readsMatch->readsLen[i], &oneResult);
	result[i] = oneResult;
}

void ReadReads(const Option * opt, const RefGenome * refGenome,
		const HashTable * hashTable) {
	FILE * fout = fopen("test_result_out.txt", "wb");

	/**************************************************************************/
	ResultMatchedReads * result;
	SIZE_T size = (MAX_MAPPING_READS + 1) * sizeof(ResultMatchedReads);
	result = (ResultMatchedReads *) malloc(size);
	if (result == NULL)
		MEMORY_ALLOCATE_ERROR
	;
	memset(result, 0x00, size);

	ReadsMatch readsMatch;
	SIZE_T readsLen = ReadWholeFile(opt->readsFile, &readsMatch.strReads);
	readsMatch.readsInBits = (InBits *) malloc(
			sizeof(InBits) * (MAX_MAPPING_READS + 1));
	if (readsMatch.readsInBits == NULL)
		MEMORY_ALLOCATE_ERROR
	;
	readsMatch.readsLen = (SIZE_T *) malloc(
			sizeof(SIZE_T) * (MAX_MAPPING_READS + 1));
	if (readsMatch.readsLen == NULL)
		MEMORY_ALLOCATE_ERROR
	;

	/**************************************************************************/
	ReadsMatch d_readsMatch;
	ResultMatchedReads * d_result;
	size = (MAX_MAPPING_READS + 1) * sizeof(InBits);
	HANDLE_ERROR(hipMalloc((void **) &d_readsMatch.readsInBits, size));
	size = (MAX_MAPPING_READS + 1) * sizeof(SIZE_T);
	HANDLE_ERROR(hipMalloc((void **) &d_readsMatch.readsLen, size));

	size = (MAX_MAPPING_READS + 1) * sizeof(ResultMatchedReads);
	HANDLE_ERROR(hipMalloc((void **) &d_result, size));
	HANDLE_ERROR(hipMemset(d_result, 0x00, size));
	/**************************************************************************/

	int readID = 0;
	/* read reads from the file*/
	readsMatch.nReads = 0;
	char strRead[MAX_LINE_LEN];
	for (SIZE_T i = 0; i < readsLen; i++) {
		int len = GetLineFromString(&readsMatch.strReads[i], strRead);
		i += len;
		if (strRead[0] == '>')
			continue;

		EncodeRead(strRead, &readsMatch.readsInBits[readsMatch.nReads], len);
		readsMatch.readsLen[readsMatch.nReads] = len;
		readsMatch.nReads++;

		if (readsMatch.nReads >= MAX_MAPPING_READS) {
			HANDLE_ERROR(
					hipMemcpy(d_readsMatch.readsInBits, readsMatch.readsInBits, (MAX_MAPPING_READS + 1) * sizeof(InBits), hipMemcpyHostToDevice));
			HANDLE_ERROR(
					hipMemcpy(d_readsMatch.readsLen, readsMatch.readsLen, (MAX_MAPPING_READS + 1) * sizeof(SIZE_T), hipMemcpyHostToDevice));
			gpu_kernel<<<BLOCKS, THREADS>>>(opt, refGenome, hashTable,
					&d_readsMatch, d_result);
			//HANDLE_ERROR(
			//		hipMemcpy(result, d_result, (MAX_MAPPING_READS + 1) * sizeof(ResultMatchedReads), hipMemcpyDeviceToHost));
			//OutPutResult(fout, result, readsMatch.nReads, readID);
			readID += readsMatch.nReads;
			readsMatch.nReads = 0;
		}
	}
	if (readsMatch.nReads > 0) {
		HANDLE_ERROR(
				hipMemcpy(d_readsMatch.readsInBits, readsMatch.readsInBits, (MAX_MAPPING_READS + 1) * sizeof(InBits), hipMemcpyHostToDevice));
		HANDLE_ERROR(
				hipMemcpy(d_readsMatch.readsLen, readsMatch.readsLen, (MAX_MAPPING_READS + 1) * sizeof(SIZE_T), hipMemcpyHostToDevice));
		gpu_kernel<<<BLOCKS, THREADS>>>(opt, refGenome, hashTable,
				&d_readsMatch, d_result);
		//HANDLE_ERROR(
		//		hipMemcpy(result, d_result, (MAX_MAPPING_READS + 1) * sizeof(ResultMatchedReads), hipMemcpyDeviceToHost));
		//OutPutResult(fout, result, readsMatch.nReads, readID);
		readID += readsMatch.nReads;
		readsMatch.nReads = 0;
	}

	fclose(fout);
	hipFree(d_readsMatch.readsInBits);
	hipFree(d_readsMatch.readsLen);
	hipFree(d_result);
	free(result);
	free(readsMatch.strReads);
	free(readsMatch.readsInBits);
}

void Matching(const Option * opt, const RefGenome * refGenome,
		const HashTable * hashTable) {
	/* device variables memory allocation start*/

	RefGenome d_refGenome;
	HashTable d_hashTable;

	/* CUDA memory allocation start*/
	d_refGenome.nRefSize = refGenome->nRefSize;
	d_refGenome.nRefSizeInWordSize = refGenome->nRefSizeInWordSize;
	SIZE_T size = refGenome->nRefSizeInWordSize * sizeof(InBits);
	HANDLE_ERROR(hipMalloc((void **) &d_refGenome.refInBits, size));
	HANDLE_ERROR(
			hipMemcpy(d_refGenome.refInBits, refGenome->refInBits, size, hipMemcpyHostToDevice));

	/**************************************************************************/
	d_hashTable.NO_OF_BUCKET = hashTable->NO_OF_BUCKET;
	size = (hashTable->NO_OF_BUCKET + 1) * sizeof(SIZE_T);
	HANDLE_ERROR(hipMalloc((void **) &d_hashTable.counter, size));
	HANDLE_ERROR(
			hipMemcpy(d_hashTable.counter, hashTable->counter, size, hipMemcpyHostToDevice));

	size = (refGenome->nRefSize + 1) * sizeof(SIZE_T);
	HANDLE_ERROR(hipMalloc((void **) &d_hashTable.index, size));
	HANDLE_ERROR(
			hipMemcpy(d_hashTable.index, hashTable->index, size, hipMemcpyHostToDevice));

	/**************************************************************************/
	ReadReads(opt, &d_refGenome, &d_hashTable);
	hipFree(d_hashTable.counter);
	hipFree(d_hashTable.index);
	hipFree(d_refGenome.refInBits);
}
