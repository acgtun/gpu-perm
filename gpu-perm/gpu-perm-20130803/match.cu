#include "match.h"
#include "seed.h"

__device__ InBits GetShiftRead(SIZE_T shift, InBits read) {
	InBits r;
	r.ub = read.ub >> shift;
	r.lb = read.lb >> shift;
	return r;
}
__device__ int STRCMP(const char * str1, int len1, const char * str2, int len2) {
	int i = 0;
	while (i < len1 && i < len2) {
		if (str1[i] < str2[i])
			return -1;
		else if (str1[i] > str2[i])
			return 1;
		i++;
	}
	if (i == len1 && i == len2)
		return 0;
	if (i == len2)
		return 1;
	else
		return 0; //if len2 > len1, the string equals to certain part of genome
}

__device__ int CMP(char * strRead, int len, SIZE_T index, const CReference * refGenome) {
	InBits r, v;
	SIZE_T s = GetKmer(refGenome, index, wordSize, &r);
	int ss = GetF2SeedForBits(r, s, &v);

	char strRef64[MAX_READ_LEN];
	DecodeReadReverse(strRef64, ss, &v);

	return STRCMP(strRead, len, strRef64, ss);
}

__device__ SIZE_T LowerBound(SIZE_T low, SIZE_T high, char * strRead, int s, const CReference * refGenome, const CHashTable * hashTable) {
	SIZE_T mid = 0;
	while (low < high) {
		mid = (low + high) / 2;
		if (CMP(strRead, s, hashTable->index[mid], refGenome) <= 0)
			high = mid;
		else
			low = mid + 1;
	}
	return low;
}

__device__ SIZE_T UpperBound(SIZE_T low, SIZE_T high, char * strRead, int s, const CReference * refGenome, const CHashTable * hashTable) {
	SIZE_T mid = 0;
	while (low < high) {
		mid = (low + high + 1) / 2;
		if (CMP(strRead, s, hashTable->index[mid], refGenome) >= 0)
			low = mid;
		else
			high = mid - 1;
	}
	return low;
}

__device__ void reverseCompliment(char * strRead, InBits readInBits, int len) {
	DecodeReadReverse(strRead, len, &readInBits);
	for (int i = 0; i < len; i++) {
		strRead[i] = complimentBase(strRead[i]);
	}
}

__device__ int ResultExist(const CResult * oneResult, SIZE_T index) {
	for (SIZE_T i = 0; i < oneResult->nRet; i++) {
		if (oneResult->nStartPos[i] == index)
			return 1;
	}
	return 0;
}

__device__ void Mapping(const MapOpt mapOpt, const CReference * refGenome, const CHashTable * hashTable, CRead oneRead,
		CResult * oneResult) {
	for (int i = 0; i <= NUMBER_OF_SHIFT; i++) {
		InBits read = GetShiftRead(i, oneRead.readInBits);
		SIZE_T hashValue = GetHashValue(read);
		InBits ret;
		int len = oneRead.readLen - i;
		int s = GetF2SeedForBits(read, len, &ret);
		char strRead[MAX_READ_LEN];
		DecodeReadReverse(strRead, s, &ret);

		SIZE_T l = hashTable->counter[hashValue];
		SIZE_T u = hashTable->counter[hashValue + 1] - 1;

		SIZE_T lower = LowerBound(l, u, strRead, s, refGenome, hashTable);
		SIZE_T upper = UpperBound(l, u, strRead, s, refGenome, hashTable);

		for (SIZE_T j = lower; j <= upper; j++) {
			if (ResultExist(oneResult, hashTable->index[j] - i))
				continue;

			int s = GetKmer(refGenome, hashTable->index[j] - i, oneRead.readLen, &ret);
			if (s != oneRead.readLen)
				continue;
			if (oneResult->nRet >= 200) {
				printf("Array touch the Boundary!\n");
				break;
			}

			SIZE_T nDiff = bitsStrNCompare(ret, oneRead.readInBits, oneRead.readLen);
			if (nDiff <= mapOpt.nMaxMismatch) {
				oneResult->nMismatch[oneResult->nRet] = nDiff;
				oneResult->nStartPos[oneResult->nRet] = hashTable->index[j] - i;
				oneResult->nRet++;
			}
		}
	}
}

__device__ void MappingOneRead(const MapOpt mapOpt, const CReference * refGenome, const CHashTable * hashTable, CRead oneRead,
		CResult * oneResult) {
	oneResult->nRet = 0;
	Mapping(mapOpt, refGenome, hashTable, oneRead, oneResult);
	char strRead[MAX_READ_LEN];
	reverseCompliment(strRead, oneRead.readInBits, oneRead.readLen);
	InBits readInBits_rev;
	EncodeRead(strRead, &readInBits_rev, oneRead.readLen);
	oneRead.readInBits = readInBits_rev;
	Mapping(mapOpt, refGenome, hashTable, oneRead, oneResult);
}
