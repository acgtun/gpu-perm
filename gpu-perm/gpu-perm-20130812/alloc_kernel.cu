#include "hip/hip_runtime.h"
#include "alloc_kernel.h"

__global__ void gpu_kernel(const MapOpt mapOpt, const CReference refGenome, const CHashTable hashTable, CReadArray reads,
		CResult * result) {
	SIZE_T i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i >= reads.nReadsNum)
		return;
	MappingOneRead(mapOpt, &refGenome, &hashTable, reads.reads[i], &(result[i]));
}

void run_kernel(FILE * fout, const MapOpt & mapOopt, const CReference * refGenome, const CHashTable * hashTable, const CReadArray * reads,
		CReadArray * d_reads, CResult * result, CResult * d_result, const SIZE_T & readID) {
	d_reads->nReadsNum = reads->nReadsNum;
	HANDLE_ERROR(hipMemcpy(d_reads->reads, reads->reads, reads->nReadsNum * sizeof(CRead), hipMemcpyHostToDevice));
	gpu_kernel<<<BLOCKS, THREADS>>>(mapOopt, *refGenome, *hashTable, *d_reads, d_result);
	HANDLE_ERROR(hipMemcpy(result, d_result, reads->nReadsNum * sizeof(CResult), hipMemcpyDeviceToHost));
	OutPutResult(fout, result, reads->nReadsNum, readID);
}

void ReadReads(const Option & opt, const CReference * refGenome, const CHashTable * hashTable) {
	FILE * fout = fopen(opt.outputFile, "wb");
	CReadArray reads, d_reads;
	MEMORY_ALLOCATE_CHECK(reads.reads = (CRead *) malloc(sizeof(CRead) * MAX_MAPPING_READS));
	HANDLE_ERROR(hipMalloc((void **) &d_reads.reads, MAX_MAPPING_READS * sizeof(CRead)));
	/************************************************************************************/
	CResult * result, *d_result;
	HANDLE_ERROR(hipMalloc((void **) &d_result, MAX_MAPPING_READS * sizeof(CResult)));
	MEMORY_ALLOCATE_CHECK(result = (CResult *) malloc(MAX_MAPPING_READS * sizeof(CResult)));
	memset(result, 0x00, MAX_MAPPING_READS * sizeof(CResult));
	/************************************************************************************/
	/* read reads from the file*/
	char * strReads;
	SIZE_T readsLen = ReadWholeFile(opt.readsFile, &strReads);

	char strRead[MAX_LINE_LEN];
	reads.nReadsNum = 0;
	SIZE_T readID = 0;
	for (SIZE_T i = 0; i < readsLen; i++) {
		SIZE_T len = GetLineFromString(&strReads[i], strRead);
		i += len;
		if (strRead[0] == '>')
			continue;

		EncodeRead(strRead, &reads.reads[reads.nReadsNum].readInBits, len);
		reads.reads[reads.nReadsNum].readLen = len;
		reads.nReadsNum++;

		if (reads.nReadsNum >= MAX_MAPPING_READS || (reads.nReadsNum > 0 && i == readsLen - 1)) {
			run_kernel(fout, opt.mapOpt, refGenome, hashTable, &reads, &d_reads, result, d_result, readID);
			readID += reads.nReadsNum;
			reads.nReadsNum = 0;
		}
	}

	fclose(fout);
	free(reads.reads);
	hipFree(d_reads.reads);
	free(strReads);
}

void Matching(const Option & opt, const CReference * refGenome, const CHashTable * hashTable) {
	CReference d_refGenome;
	CHashTable d_hashTable;

	SIZE_T sizeCounter = sizeof(SIZE_T) * hashTable->nSizeCounter;
	HANDLE_ERROR(hipMalloc((void ** )&(d_hashTable.counter), sizeCounter));
	HANDLE_ERROR(hipMemcpy(d_hashTable.counter, hashTable->counter, sizeCounter, hipMemcpyHostToDevice));
	d_hashTable.nSizeCounter = hashTable->nSizeCounter;

	SIZE_T sizeIndex = sizeof(SIZE_T) * hashTable->nSizeIndex;
	HANDLE_ERROR(hipMalloc((void ** )&(d_hashTable.index), sizeIndex));
	HANDLE_ERROR(hipMemcpy(d_hashTable.index, hashTable->index, sizeIndex, hipMemcpyHostToDevice));
	d_hashTable.nSizeIndex = hashTable->nSizeIndex;

	SIZE_T sizeRef = sizeof(InBits) * refGenome->nRefSizeInWordSize;
	HANDLE_ERROR(hipMalloc((void ** )&(d_refGenome.refInBits), sizeRef));
	HANDLE_ERROR(hipMemcpy(d_refGenome.refInBits, refGenome->refInBits, sizeRef, hipMemcpyHostToDevice));
	d_refGenome.nRefSize = refGenome->nRefSize;
	d_refGenome.nRefSizeInWordSize = refGenome->nRefSizeInWordSize;

	ReadReads(opt, &d_refGenome, &d_hashTable);

	/* free memory*/
	free(refGenome->refInBits);
	free(hashTable->counter);
	free(hashTable->index);
	hipFree(d_refGenome.refInBits);
	hipFree(d_hashTable.counter);
	hipFree(d_hashTable.index);
}
