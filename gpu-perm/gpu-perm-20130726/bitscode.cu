#include "bitscode.h"

/*
 * uiReadLength must < WORD_SIZE
 * WORD_SIZE is 32 bp in 32 big machine and 64 bp in 64 bit machine
 * Each base is encoded into 2 bits: A -> 00, C->01, G->10 and T->11.
 * These two digits are located in two word, for bits operation.
 * The first nucleotide is encoded as the last digit.
 */

__device__ __host__ void EncodeRead(const char * strRead, InBits * readsInBits,
		int len) {

	readsInBits->ub = 0;
	readsInBits->lb = 0;

	// A 00
	// C 01
	// G 10
	// T 11

	for (int i = len - 1; i >= 0; i--) {
		if (strRead[i] == 'A' || strRead[i] == 'a') {

		} else if (strRead[i] == 'C' || strRead[i] == 'c') {
			readsInBits->lb++;
		} else if (strRead[i] == 'G' || strRead[i] == 'g') {
			readsInBits->ub++;
		} else if (strRead[i] == 'T' || strRead[i] == 't') {
			readsInBits->ub++;
			readsInBits->lb++;
		} else {
			//printf("Not A, C, G, T\n");
			//anything else as A
		}
		if (i != 0) {
			readsInBits->ub <<= 1; //left shift 1
			readsInBits->lb <<= 1;
		}
	}
}

void printWORD(WORD_SIZE word, SIZE_T len) {
	printf("\n");
	for (SIZE_T i = 0; i < len; i++) {
		if ((word & 0x01) == 1) {
			printf("1");
		} else {
			printf("0");
		}
		word >>= 1;
	}
	printf("\n");
}

void printWORD2File(FILE * fout, WORD_SIZE word, SIZE_T len) {
	for (SIZE_T i = 0; i < len; i++) {
		if ((word & 0x01) == 1) {
			fprintf(fout, "1");
		} else {
			fprintf(fout, "0");
		}
		word >>= 1;
	}
	fprintf(fout, "\n");
}

__device__ __host__ void DecodeRead(char * strReads, int readLen,
		const InBits * readsInBits) {
	WORD_SIZE UpperBits = readsInBits->ub;
	WORD_SIZE LowerBits = readsInBits->lb;
	int strReadsl = 0;
	for (int i = 0; i < readLen; i++) {
		WORD_SIZE c = (UpperBits & 0x01) << 1 | (LowerBits & 0x01);
		switch (c) {
		case 0x00:
			strReads[strReadsl++] = 'A';
			break;
		case 0x01:
			strReads[strReadsl++] = 'C';
			break;
		case 0x02:
			strReads[strReadsl++] = 'G';
			break;
		case 0x03:
			strReads[strReadsl++] = 'T';
			break;
		default:
			strReads[strReadsl++] = 'N';
			break;
		}
		LowerBits >>= 1;
		UpperBits >>= 1;
	}
	strReads[strReadsl] = 0;
}

__device__ __host__ void Swap(char * strVal, int len) {
	char chr;
	for (int i = 0; i < len / 2; i++) {
		chr = strVal[i];
		strVal[i] = strVal[len - i - 1];
		strVal[len - i - 1] = chr;
	}
}

__device__ __host__ void DecodeReadReverse(char * strRead, int readLen,
		const InBits * readsInBits) {
	DecodeRead(strRead, readLen, readsInBits);
	Swap(strRead, readLen);
}
