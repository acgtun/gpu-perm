#include "hip/hip_runtime.h"
#include "alloc_kernel.h"

__global__ void gpu_kernel(Option opt, const RefGenome * refGenome,
		const HashTable * hashTable, ReadsMatch * readsMatch,
		ResultMatchedReads * result, SIZE_T nReads) {
	SIZE_T i = threadIdx.x + blockDim.x * blockIdx.x;
	//	printf("i = %d\n", i);
	//	printf("readsMatch->nReads = %d\n", readsMatch->nReads);
	if (i >= nReads)
		return;

	MappingOneRead(opt.matchOpt, refGenome, hashTable,
			readsMatch->readsInBits[i], readsMatch->readsLen[i], &result[i]);
}

void ReadReads(const Option * opt, const RefGenome * refGenome,
		const HashTable * hashTable) {
	//FILE * fout = fopen("test_result_out.txt", "wb");

	/**************************************************************************/
	ResultMatchedReads * result;
	SIZE_T size = (MAX_MAPPING_READS + 1) * sizeof(ResultMatchedReads);
	result = (ResultMatchedReads *) malloc(size);
	if (result == NULL)
		MEMORY_ALLOCATE_ERROR
			;
	memset(result, 0x00, size);

	ReadsMatch readsMatch;
	SIZE_T readsLen = ReadWholeFile(opt->readsFile, &readsMatch.strReads);
	readsMatch.readsInBits = (InBits *) malloc(
			sizeof(InBits) * (MAX_MAPPING_READS + 1));
	if (readsMatch.readsInBits == NULL)
		MEMORY_ALLOCATE_ERROR
			;
	readsMatch.readsLen = (SIZE_T *) malloc(
			sizeof(SIZE_T) * (MAX_MAPPING_READS + 1));
	if (readsMatch.readsLen == NULL)
		MEMORY_ALLOCATE_ERROR
			;

	/**************************************************************************/
	ReadsMatch * d_readsMatch;
	HANDLE_ERROR(hipMalloc((void **) &d_readsMatch, sizeof(ReadsMatch)));
	size = (MAX_MAPPING_READS + 1) * sizeof(InBits);
	HANDLE_ERROR(hipMalloc((void **) &(d_readsMatch->readsInBits), size));
	size = (MAX_MAPPING_READS + 1) * sizeof(SIZE_T);
	HANDLE_ERROR(hipMalloc((void **) &(d_readsMatch->readsLen), size));

	ResultMatchedReads * d_result;
	size = (MAX_MAPPING_READS + 1) * sizeof(ResultMatchedReads);
	HANDLE_ERROR(hipMalloc((void **) &d_result, size));
	HANDLE_ERROR(hipMemset(d_result, 0x00, size));
	/**************************************************************************/

	int readID = 0;
	/* read reads from the file*/
	readsMatch.nReads = 0;
	char strRead[MAX_LINE_LEN];
	for (SIZE_T i = 0; i < readsLen; i++) {
		int len = GetLineFromString(&readsMatch.strReads[i], strRead);
		i += len;
		if (strRead[0] == '>')
			continue;

		EncodeRead(strRead, &readsMatch.readsInBits[readsMatch.nReads], len);
		readsMatch.readsLen[readsMatch.nReads] = len;
		readsMatch.nReads++;

		if (readsMatch.nReads >= MAX_MAPPING_READS) {
			HANDLE_ERROR(
					hipMemcpy(d_readsMatch->readsInBits, readsMatch.readsInBits, (readsMatch.nReads) * sizeof(InBits), hipMemcpyHostToDevice));
			HANDLE_ERROR(
					hipMemcpy(d_readsMatch->readsLen, readsMatch.readsLen, (readsMatch.nReads) * sizeof(SIZE_T), hipMemcpyHostToDevice));
			gpu_kernel<<<BLOCKS, THREADS>>>(*opt, refGenome, hashTable,
					d_readsMatch, d_result, readsMatch.nReads);
			//HANDLE_ERROR(
			//		hipMemcpy(result, d_result, (MAX_MAPPING_READS + 1) * sizeof(ResultMatchedReads), hipMemcpyDeviceToHost));
			//OutPutResult(fout, result, readsMatch.nReads, readID);
			readID += readsMatch.nReads;
			readsMatch.nReads = 0;
		}
	}
	if (readsMatch.nReads > 0) {
		HANDLE_ERROR(
				hipMemcpy(d_readsMatch->readsInBits, readsMatch.readsInBits, (readsMatch.nReads) * sizeof(InBits), hipMemcpyHostToDevice));
		HANDLE_ERROR(
				hipMemcpy(d_readsMatch->readsLen, readsMatch.readsLen, (readsMatch.nReads) * sizeof(SIZE_T), hipMemcpyHostToDevice));
		gpu_kernel<<<BLOCKS, THREADS>>>(*opt, refGenome, hashTable,
				d_readsMatch, d_result, readsMatch.nReads);
		//HANDLE_ERROR(
		//		hipMemcpy(result, d_result, (MAX_MAPPING_READS + 1) * sizeof(ResultMatchedReads), hipMemcpyDeviceToHost));
		//OutPutResult(fout, result, readsMatch.nReads, readID);
		readID += readsMatch.nReads;
		readsMatch.nReads = 0;
	}

	//fclose(fout);
	hipFree(d_readsMatch->readsInBits);
	hipFree(d_readsMatch->readsLen);
	hipFree(d_readsMatch);
	hipFree(d_result);
	free(result);
	free(readsMatch.strReads);
	free(readsMatch.readsInBits);
	free(readsMatch.readsLen);
}

void Matching(const Option * opt, const RefGenome * refGenome,
		const HashTable * hashTable) {
	FILE * fout = fopen("see.txt", "wb");
	fprintf(fout, "jinru\n");
	/* device variables memory allocation start*/
	
	RefGenome * d_refGenome;
	HashTable * d_hashTable;
	
	//fclose(fout);
	/* CUDA memory allocation start*/
	HANDLE_ERROR(hipMalloc((void **) &d_refGenome, sizeof(d_refGenome)));
	HANDLE_ERROR(hipMalloc((void **) &d_hashTable, sizeof(d_hashTable)));
	
	fprintf(fout, "%d %d\n", *refGenome->nRefSize, *refGenome->nRefSizeInWordSize);
	fprintf(fout, "see2\n");
	fclose(fout);
	/**************************************************************************/
	HANDLE_ERROR(hipMalloc((void **) &(d_refGenome->nRefSize), sizeof(SIZE_T)));

	FILE * fsee = fopen("see2.txt", "wb");
	fprintf(fsee, "see2.txt");
	fclose(fsee);

	HANDLE_ERROR(
			hipMalloc((void **) &(d_refGenome->nRefSizeInWordSize), sizeof(SIZE_T)));
	//fclose(fout);
	HANDLE_ERROR(
			hipMemcpy(d_refGenome->nRefSize, refGenome->nRefSize, sizeof(SIZE_T), hipMemcpyHostToDevice));
	HANDLE_ERROR(
			hipMemcpy(d_refGenome->nRefSizeInWordSize, refGenome->nRefSizeInWordSize, sizeof(SIZE_T), hipMemcpyHostToDevice));
	//fprintf(fout, "here\n");
	//fclose(fout);
	LOG_INFO
	/**************************************************************************/
	SIZE_T size = (*(refGenome->nRefSizeInWordSize) + 1) * sizeof(InBits);
	HANDLE_ERROR(hipMalloc((void **) &(d_refGenome->refInBits), size));
	HANDLE_ERROR(
			hipMemcpy(d_refGenome->refInBits, refGenome->refInBits, size, hipMemcpyHostToDevice));

	/**************************************************************************/
	//d_hashTable.NO_OF_BUCKET = hashTable->NO_OF_BUCKET;
	size = (NO_OF_BUCKET + 1) * sizeof(SIZE_T);
	HANDLE_ERROR(hipMalloc((void **) &(d_hashTable->counter), size));
	HANDLE_ERROR(
			hipMemcpy(d_hashTable->counter, hashTable->counter, size, hipMemcpyHostToDevice));

	size = (*(refGenome->nRefSize) + 1) * sizeof(SIZE_T);
	HANDLE_ERROR(hipMalloc((void **) &(d_hashTable->index), size));
	HANDLE_ERROR(
			hipMemcpy(d_hashTable->index, hashTable->index, size, hipMemcpyHostToDevice));

	/**************************************************************************/
	ReadReads(opt, d_refGenome, d_hashTable);

	hipFree(d_refGenome->nRefSize);
	hipFree(d_refGenome->nRefSizeInWordSize);
	hipFree(d_hashTable->counter);
	hipFree(d_hashTable->index);
	hipFree(d_refGenome->refInBits);
	hipFree(d_refGenome);
	hipFree(d_hashTable);

	free(refGenome->refInBits);
	free(refGenome->nRefSize);
	free(refGenome->nRefSizeInWordSize);
	free(hashTable->counter);
	free(hashTable->index);
}
